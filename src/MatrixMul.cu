#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/MatrixMul.cuh"

__global__
void MatrixMul(int heightA, int widthA, int widthB, float *matrixA, float *matrixB, float *matrixResult) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < heightA && col < widthB) {
        float value = 0.0;

        for (int k = 0; k < widthA; ++k) {
            value += matrixA[row * widthA + k] * matrixB[k * widthB + col];
        }

        matrixResult[row * widthB + col] = value;
    }
}

int main() {
    int heightA = 2, widthA = 3, widthB = 2;

    // Host matrices
    float h_matrixA[] = {1, 2, 3, 4, 5, 6};
    float h_matrixB[] = {7, 8, 9, 10, 11, 12};
    float h_matrixResult[4] = {0};

    float *d_matrixA, *d_matrixB, *d_matrixResult;

    hipMalloc(&d_matrixA, heightA * widthA * sizeof(float));
    hipMalloc(&d_matrixB, widthA * widthB * sizeof(float));
    hipMalloc(&d_matrixResult, heightA * widthB * sizeof(float));

    hipMemcpy(d_matrixA, h_matrixA, heightA * widthA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, h_matrixB, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((widthB + blockDim.x - 1) / blockDim.x, (heightA + blockDim.y - 1) / blockDim.y);

    MatrixMul<<<gridDim, blockDim>>>(heightA, widthA, widthB, d_matrixA, d_matrixB, d_matrixResult);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_matrixResult, d_matrixResult, heightA * widthB * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < heightA; ++i) {
        for (int j = 0; j < widthB; ++j) {
            printf("%f ", h_matrixResult[i * widthB + j]);
        }
        printf("\n");
    }

    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixResult);

    return 0;
}
